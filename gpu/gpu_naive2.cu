
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define BLOCK_DIM 8192 

__device__ bool isPrime(int number){
  int i;
  for( i = 2; i < number;i++){
    if (number%i == 0)
        return  false;
  }
  return true;
}

__global__ void primality(bool* result,int number){
  int i = threadIdx.x + blockIdx.x * blockDim.x +2;
  while( i < number){
      result[i] = isPrime(i);
      i+= BLOCK_DIM;
  }
}

int main(int argc,char* argv[]){
  int value = atoi(argv[1]);
  bool *numbers;

  bool *results = (bool*)malloc(sizeof(bool)*value);
  hipMalloc( (void**)&numbers,sizeof(bool)*value); 

  dim3  blocks_per_grid(64,1);
  dim3  thread_per_block(128,1);

  primality<<<blocks_per_grid,thread_per_block>>>(numbers,value);
  hipError_t Error = hipGetLastError();
  if( hipSuccess != Error){
        printf("CUDA Error en el Llamado del Kernel: %s\n", hipGetErrorString(Error));
  }
  
  hipMemcpy(results,numbers,sizeof(bool)*value,hipMemcpyDeviceToHost); 
  
  Error = hipGetLastError();
  if( hipSuccess != Error){
        printf("CUDA Error en la copia: %s\n", hipGetErrorString(Error));
  }
  hipFree(numbers);
  
  int i = 2;
  int counter = 0;
  for( ; i < value;i++)
      if (results[i])
        counter++;
  printf("%d\n",counter);

  free(results);

  return 0;
}

