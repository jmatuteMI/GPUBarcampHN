
#include <hip/hip_runtime.h>

__global__ void add(int a,int b,int *c){
  *c = a+b;
}

int main(){
  int c;
  int *resp;

  hipMalloc( (void**)&resp,sizeof(int));

  add<<<1,1>>>(2,3,resp);

  hipMemcpy(&c,resp,sizeof(int),hipMemcpyDeviceToHost);

  hipFree(resp);

  return c;
}
