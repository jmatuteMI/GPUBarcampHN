#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#define BLOCK_DIM 8192

__global__ void primality(int* result,int val,int limit,int offset){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int start = i*val + offset;
  while (start < limit){
    if(result[start] % val == 0){
       result[start] = 0;
    }
    i += BLOCK_DIM;
    start += i*val + offset; 
  } 
}

int main(int argc,char *argv[]){
  int value = atoi(argv[1]);
  int* numbers = (int*)malloc(sizeof(int)*(value));
  int i;
  int trying,squared;
  numbers[0] = numbers[1] = 0;
  for(i = 2; i < value;i++)
      numbers[i] = i;
  
  int start;
  squared = 0;
  start = -1; 
  dim3  blocks_per_grid(64,1);
  dim3  thread_per_block(128,1);
  
  int *cuda_mem_array;
  hipMalloc( (void**)&cuda_mem_array,(value)*sizeof(int));
  hipMemcpy( cuda_mem_array ,numbers,value*sizeof(int),hipMemcpyHostToDevice);
  hipError_t Error = hipGetLastError();
  if( hipSuccess != Error){
        printf("CUDA Error: %s\n", hipGetErrorString(Error));
  }
  while (1){
     while( numbers[++start] == 0 );
     squared = start*start;
     if (squared > value)
        break;
     trying = numbers[start];
     primality<<<blocks_per_grid,thread_per_block>>>(cuda_mem_array,trying,value,squared);
     hipDeviceSynchronize();
     hipMemcpy(numbers,cuda_mem_array,value*sizeof(int),hipMemcpyDeviceToHost); 
     Error = hipGetLastError();
     if( hipSuccess != Error){
        printf("CUDA Error: %s\n", hipGetErrorString(Error));
     }

     hipDeviceSynchronize();
  }

  hipFree(cuda_mem_array);

  int total = 0;
  for( i = 0; i < value;i++)
    if ( numbers[i] != 0 ) total += 1;

  free(numbers);
  printf("%d\n",total);
  return 0;  
}

